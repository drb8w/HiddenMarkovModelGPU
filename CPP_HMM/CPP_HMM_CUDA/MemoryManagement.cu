#include "MemoryManagement.cuh"

#include <stdio.h>
#include <stdlib.h> 
#include <string.h>

ComputationEnvironment glob_Env = ComputationEnvironment::GPU;

__host__ hipError_t allocateDeviceVector(IntHdl pVector, int numberOfElements)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		break;
	case ComputationEnvironment::CPU:
		*pVector = (int *)malloc(numberOfElements * sizeof(int));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t allocateDeviceVector(FloatHdl pVector, int numberOfElements)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		break;
	case ComputationEnvironment::CPU:
		*pVector = (float *)malloc(numberOfElements * sizeof(float));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t allocateDeviceVector(DoubleHdl pVector, int numberOfElements)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		break;
	case ComputationEnvironment::CPU:
		*pVector = (double *)malloc(numberOfElements * sizeof(double));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(IntPtr dst, const IntPtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(int), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		memccpy(dst, src, numberOfElements, sizeof(int));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(FloatPtr dst, const FloatPtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(float), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		memccpy(dst, src, numberOfElements, sizeof(float));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(DoublePtr dst, const DoublePtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(double), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		memccpy(dst, src, numberOfElements, sizeof(double));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t deviceFree(void *devPtr)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipFree(devPtr);
		break;
	case ComputationEnvironment::CPU:
		free(devPtr);
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}
