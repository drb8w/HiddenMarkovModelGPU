#pragma once

#include "hip/hip_runtime.h"
#include ""
#include "MemoryManagement.cuh"
#include "Utilities.h"
#include "VectorMath.cuh"

#include <stdio.h>
#include <cmath>
#include <fstream>
#include <iostream>
using namespace std;

// ------------------------------------------------------------------------------------------------------
// global states
// ------------------------------------------------------------------------------------------------------
extern ComputationEnvironment glob_Env;

int main(int argc, char* argv[])
{

	// dim 3x4 ( r x c )
	double A[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12 };

	// dim 4x3 ( r x c )
	double B[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12 };

	// result matrix ; dim 3x3
	double *C = nullptr;
	glob_Env = ComputationEnvironment::CPU;
	allocateDeviceVector(&C, 9, true);
	glob_Env = ComputationEnvironment::GPU;

	double *dev_A = nullptr;
	double *dev_B = nullptr;
	double *dev_C = nullptr;

	allocateDeviceVector(&dev_A, 12,true);
	allocateDeviceVector(&dev_B, 12,true);
	allocateDeviceVector(&dev_C, 9,true);

	double* A_start = &A[0];
	double* B_start = &B[0];

	memcpyVector(dev_A, A_start, 12, hipMemcpyHostToDevice);
	memcpyVector(dev_B, B_start, 12, hipMemcpyHostToDevice);

	cublasMultiplyDouble(3, 3, 4, dev_A, dev_B, dev_C);

	memcpyVector(C, dev_C, 9, hipMemcpyDeviceToHost);
	memcpyVector(A, dev_A, 12, hipMemcpyDeviceToHost);
	memcpyVector(B, dev_B, 12, hipMemcpyDeviceToHost);

	deviceFree(dev_A);
	deviceFree(dev_B);
	deviceFree(dev_C);

	/*
	 * 70  80  90
	 * 158 184 210
	 * 246 288 330
	 */

	for (int i = 0; i < 9; i++)
	{
		cout << C[i] << " ";
	}



}