#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "MemoryManagement.cuh"

#include "Matricies.h"
#include "Observation.h"

#include <stdio.h>
#include <cmath>
#include <fstream>
#include <iostream>
using namespace std;

// ------------------------------------------------------------------------------------------------------
// global states
// ------------------------------------------------------------------------------------------------------
extern ComputationEnvironment glob_Env;

// ------------------------------------------------------------------------------------------------------
// forward declarations
// ------------------------------------------------------------------------------------------------------

__global__ void fwKernel(double *p, const double *transition, const double *emission, int obs);
__global__ void forwardKernel(double *dev_Alpha_trelis_2D, double *dev_probs_3D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int T_noOfObservations, int idx_obs);

__host__ hipError_t ForwardAlgorithm(const double *dev_Pi_startProbs_1D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int T_noOfObservations, double *dev_Alpha_trelis_2D, double *dev_probs_3D, double &likelyhood);
__host__ hipError_t ForwardAlgorithmGPU(const double *dev_Pi_startProbs_1D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int T_noOfObservations, double *dev_Alpha_trelis_2D, double *dev_probs_3D, double &likelyhood);
__host__ hipError_t ForwardAlgorithmCPU(const double *dev_Pi_startProbs_1D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int T_noOfObservations, double *dev_Alpha_trelis_2D, double *dev_probs_3D, double &likelyhood);

// ------------------------------------------------------------------------------------------------------


int main(int argc, char* argv[])
{

	cout << "start...\n";

	hipError_t cudaStatus;
	double *dev_A_stateTransProbs_2D = nullptr;
	double *dev_B_obsEmissionProbs_2D = nullptr;
	double *dev_probs_3D = nullptr;
	double *dev_Pi_startProbs_1D = nullptr;
	int *dev_O_obsSequence_1D = nullptr;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	Matricies* matricies = new Matricies();
	Observation* observations = new Observation();
	int N_noOfStates = matricies->N;
	int V_noOfObsSymbols = matricies->V;

	matricies->loadMatricies(argv[1]);
	observations->loadObservations(argv[1]);

	// --------------------------------------------------------------------------------------------------------

	if ((cudaStatus = allocateDeviceVector(&dev_Pi_startProbs_1D, N_noOfStates)) != hipSuccess) {
		return cudaStatus;
	}

	if ((cudaStatus = allocateDeviceVector(&dev_A_stateTransProbs_2D, N_noOfStates*N_noOfStates)) != hipSuccess) {
		deviceFree(dev_Pi_startProbs_1D);
		return cudaStatus;
	}
	
	if ((cudaStatus = allocateDeviceVector(&dev_B_obsEmissionProbs_2D, N_noOfStates*V_noOfObsSymbols)) != hipSuccess) {
		deviceFree(dev_Pi_startProbs_1D); 
		deviceFree(dev_A_stateTransProbs_2D);
		return cudaStatus;
	}

	// --------------------------------------------------------------------------------------------------------
	// Copy input vectors from host memory to GPU buffers.
	if ((cudaStatus = memcpyVector(dev_Pi_startProbs_1D, matricies->piAsArray(), N_noOfStates, hipMemcpyHostToDevice)) != hipSuccess) {
		deviceFree(dev_Pi_startProbs_1D);
		deviceFree(dev_A_stateTransProbs_2D);
		deviceFree(dev_B_obsEmissionProbs_2D);
		return cudaStatus;
	}

	if ((cudaStatus = memcpyVector(dev_A_stateTransProbs_2D, matricies->transitionAsArray(), N_noOfStates*N_noOfStates, hipMemcpyHostToDevice)) != hipSuccess) {
		deviceFree(dev_Pi_startProbs_1D); 
		deviceFree(dev_A_stateTransProbs_2D);
		deviceFree(dev_B_obsEmissionProbs_2D);
		return cudaStatus;
	}

	if ((cudaStatus = memcpyVector(dev_B_obsEmissionProbs_2D, matricies->emissionAsArray(), N_noOfStates*V_noOfObsSymbols, hipMemcpyHostToDevice)) != hipSuccess) {
		deviceFree(dev_Pi_startProbs_1D); 
		deviceFree(dev_A_stateTransProbs_2D);
		deviceFree(dev_B_obsEmissionProbs_2D);
		return cudaStatus;
	}

	// --------------------------------------------------------------------------------------------------------

	vector<vector<unsigned int>*>* sequences = &observations->sequences;
	int numberOfObservations = sequences->size();

	// for each obs. sequence do
	for (unsigned int i = 0; i<numberOfObservations; i++) {

		cout << "starting fw alg for obs sequence...\n";

		vector<unsigned int>* O_obsSequence = sequences->at(i);
		int T_noOfObservations = O_obsSequence->size();

		double* host_probs_3D = new double[N_noOfStates * N_noOfStates * T_noOfObservations];

		// array to store the observation sequence
		if ((cudaStatus = allocateDeviceVector(&dev_O_obsSequence_1D, T_noOfObservations)) != hipSuccess) {
			deviceFree(dev_Pi_startProbs_1D);
			deviceFree(dev_A_stateTransProbs_2D);
			deviceFree(dev_B_obsEmissionProbs_2D);
			return cudaStatus;
		}

		// array to store all probabilities.
		if ((cudaStatus = allocateDeviceVector(&dev_probs_3D, N_noOfStates * N_noOfStates * T_noOfObservations)) != hipSuccess) {
			deviceFree(dev_Pi_startProbs_1D); 
			deviceFree(dev_A_stateTransProbs_2D);
			deviceFree(dev_B_obsEmissionProbs_2D);
			deviceFree(dev_O_obsSequence_1D);
			return cudaStatus;
		}

		// array to store the trellis
		double *dev_Alpha_trelis_2D = nullptr;
		if ((cudaStatus = allocateDeviceVector(&dev_Alpha_trelis_2D, T_noOfObservations * N_noOfStates)) != hipSuccess) {
			deviceFree(dev_Pi_startProbs_1D); 
			deviceFree(dev_A_stateTransProbs_2D);
			deviceFree(dev_B_obsEmissionProbs_2D);
			deviceFree(dev_O_obsSequence_1D);
			deviceFree(dev_probs_3D);
			deviceFree(dev_Alpha_trelis_2D);
			return cudaStatus;
		}

		// --------------------------------------------------------------------------------------------------------

		double likelyhood = 0;
		cudaStatus = ForwardAlgorithm(dev_Pi_startProbs_1D, dev_A_stateTransProbs_2D, dev_B_obsEmissionProbs_2D, dev_O_obsSequence_1D, N_noOfStates, V_noOfObsSymbols, T_noOfObservations, dev_Alpha_trelis_2D, dev_probs_3D, likelyhood);
		
		// --------------------------------------------------------------------------------------------------------

		if (cudaStatus != hipSuccess) {
			deviceFree(dev_Pi_startProbs_1D); 
			deviceFree(dev_A_stateTransProbs_2D);
			deviceFree(dev_B_obsEmissionProbs_2D);
			deviceFree(dev_O_obsSequence_1D);
			deviceFree(dev_probs_3D);
			deviceFree(dev_Alpha_trelis_2D);
			return cudaStatus;
		}

		// Copy output vector from GPU buffer to host memory.
		if ((cudaStatus = memcpyVector(host_probs_3D, dev_probs_3D, N_noOfStates * N_noOfStates * T_noOfObservations, hipMemcpyDeviceToHost)) != hipSuccess) {
			deviceFree(dev_Pi_startProbs_1D); 
			deviceFree(dev_A_stateTransProbs_2D);
			deviceFree(dev_B_obsEmissionProbs_2D);
			deviceFree(dev_O_obsSequence_1D);
			deviceFree(dev_probs_3D);
			deviceFree(dev_Alpha_trelis_2D);
			return cudaStatus;
		}

		delete[] host_probs_3D;
		deviceFree(dev_probs_3D);
		deviceFree(dev_Alpha_trelis_2D);

	}

	deviceFree(dev_Pi_startProbs_1D);
	deviceFree(dev_A_stateTransProbs_2D);
	deviceFree(dev_B_obsEmissionProbs_2D);

	cout << "end\n";

	return 0;
}

// ------------------------------------------------------------------------------------------------------

__global__ void forwardKernel(double *dev_Alpha_trelis_2D, double *dev_probs_3D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int T_noOfObservations, int idx_obs)
{
	// ------------------------------------------------------------------------------------------------------
	// Indexing for 2D-Grid, but called as 1D-Grid
	// ------------------------------------------------------------------------------------------------------
	//int ix = blockDim.x*blockIdx.x + threadIdx.x; // i
	//int iy = blockDim.y*blockIdx.y + threadIdx.y; // j

	//int idx_trans = iy * blockDim.x + ix; // blockDim.x == blockDim.y, cuda_2.pdf s.31
	//int idx_emit = ix * blockDim.x + obs;
	//int idx_prob = blockDim.x * blockDim.y * obs + blockDim.x * ix + iy;

	// ------------------------------------------------------------------------------------------------------
	// Indexing for 1D-Grid, called as 1D-Grid
	// ------------------------------------------------------------------------------------------------------
	// reference implementation: int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// vector layout: (i,j,t)

	int i = threadIdx.x;
	int j = blockIdx.x;
	int t = idx_obs;
	int dim1 = blockDim.x;
	int dim2 = gridDim.x;
	int dim3 = T_noOfObservations; // size of observation sequence

	// calculate transition and emmision index in 2D transition and emmision arrays of size dim1 * dim2:
	// a_ji
	int idx_a_ji = j + i*dim1;
	// b_it
	int idx_b_it = i + t*dim1;
	// calculate probability index of 3D probability array of size dim1 * dim2 * dim3:
	// p = a_ji * b_it ... only temporary value, maybe p_jit ???
	int idx_p = j + i*dim1 + t*dim1*dim2;
	// calculate alpha index of 2D trellis array of size dim1 * dim3:
	// alpha_ti = alpha_ti + alpha_(t-1)j * p
	int idx_alpha_ti = t * i*dim3;
	int idx_alpha_tm1j = (t-1) * j*dim3;
	// ------------------------------------------------------------------------------------------------------

	double a_ji = dev_A_stateTransProbs_2D[idx_a_ji];
	double b_it = dev_B_obsEmissionProbs_2D[idx_b_it];
	double p = a_ji * b_it;
	dev_probs_3D[idx_p] = p;
	dev_Alpha_trelis_2D[idx_alpha_ti] = dev_Alpha_trelis_2D[idx_alpha_ti] + dev_Alpha_trelis_2D[idx_alpha_tm1j] * p;
}

__global__ void fwKernel(double *p, const double *transition, const double *emission, int obs){

	int ix = blockDim.x*blockIdx.x + threadIdx.x; // i
	int iy = blockDim.y*blockIdx.y + threadIdx.y; // j

	int idx_trans = iy * blockDim.x + ix; // blockDim.x == blockDim.y, cuda_2.pdf s.31
	int idx_emit = ix * blockDim.x + obs;
	int idx_prob = blockDim.x * blockDim.y * obs + blockDim.x * ix + iy;

	double trans = transition[idx_trans];
	double emis = emission[idx_emit];
	p[idx_prob] = trans * emis;


}

// ------------------------------------------------------------------------------------------------------
// wrapper functions to switch transparently between GPU and CPU calcuation 
// without changing the main algorithms
// ------------------------------------------------------------------------------------------------------

__host__ hipError_t ForwardAlgorithmGPU(const double *dev_Pi_startProbs_1D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int T_noOfObservations, double *dev_Alpha_trelis_2D, double *dev_probs_3D, double &likelyhood)
{
	// ------------------------------------------------------------------------------------------------------
	// Initialization of the Alpha_trelis
	// ------------------------------------------------------------------------------------------------------
	// a_0i = pi_i --- actually should be, but to be sure Pi is transported in an extra vector
	// alpha_1(i) = Pi_i*b_i(O_1)

	int startingObs = dev_O_obsSequence_1D[0];

	for (unsigned int i = 0; i < N_noOfStates; i++)
	{
		double alpha_1_i = dev_Pi_startProbs_1D[i] * dev_B_obsEmissionProbs_2D[i*V_noOfObsSymbols + startingObs];
		dev_Alpha_trelis_2D[i] = alpha_1_i; // init first row of trelis
	}

	// ------------------------------------------------------------------------------------------------------
	
	for (unsigned int idx_obs = 1; idx_obs < T_noOfObservations; idx_obs++){
		
		// call kernel for NxT matrix ops (N is the number of states, T is the number of observations)
		// Launch a kernel on the GPU with one thread for each element.
		//forwardKernel << <dim3(N_noOfStates, N_noOfStates), dim3(N_noOfStates, N_noOfStates) >> >(dev_Alpha_trelis_2D, dev_probs_3D, dev_A_stateTransProbs_2D, dev_B_obsEmissionProbs_2D, obs);
		forwardKernel << <N_noOfStates, N_noOfStates >> >(dev_Alpha_trelis_2D, dev_probs_3D, dev_A_stateTransProbs_2D, dev_B_obsEmissionProbs_2D, dev_O_obsSequence_1D, T_noOfObservations, idx_obs);

	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipError_t cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

	// ------------------------------------------------------------------------------------------------------
	// extract likelyhood as the goal of the algorithm
	// likelyhood = alpha_(Obs_T)endstate

	// get index of last obervation symbol in set of observation symbols
	int idx_obs_T = 0;
	// get index of end state in set of states
	int idx_state_end = 0;
	// get index in trellis and return as likelyhood
	int idx_alpha_obsT_stateEnd = 0;

	likelyhood = dev_Alpha_trelis_2D[idx_alpha_obsT_stateEnd];

	return cudaStatus;
}

__host__ hipError_t ForwardAlgorithmCPU(const double *dev_Pi_startProbs_1D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int T_noOfObservations, double *dev_Alpha_trelis_2D, double *dev_probs_3D, double &likelyhood)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;

	// ------------------------------------------------------------------------------------------------------
	// Initialization of the Alpha_trelis
	// ------------------------------------------------------------------------------------------------------
	// a_0i = pi_i --- actually should be, but to be sure Pi is transported in an extra vector
	// alpha_1(i) = Pi_i*b_i(O_1)

	int startingObs = dev_O_obsSequence_1D[0];

	for (unsigned int i = 0; i < N_noOfStates; i++)
	{
		double alpha_1_i = dev_Pi_startProbs_1D[i] * dev_B_obsEmissionProbs_2D[i*V_noOfObsSymbols + startingObs];
		dev_Alpha_trelis_2D[i] = alpha_1_i; // init first row of trelis
	}

	// ------------------------------------------------------------------------------------------------------


	// call kernel for NxT matrix ops (N is the number of states, T is the number of observations)
	// Launch a kernel on the GPU with one thread for each element.
	//	fwKernel << <N, N >> >(dev_probability, dev_transition, dev_emission, i);
	for (int i = 0; i < N_noOfStates; i++)
	{
		for (int j = 0; j < N_noOfStates; j++)
		{

		}
	}

	return cudaStatus;
}

__host__ hipError_t ForwardAlgorithm(const double *dev_Pi_startProbs_1D, const double *dev_A_stateTransProbs_2D, const double *dev_B_obsEmissionProbs_2D, const int *dev_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int T_noOfObservations, double *dev_Alpha_trelis_2D, double *dev_probs_3D, double &likelyhood)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;

	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = ForwardAlgorithmGPU(dev_Pi_startProbs_1D, dev_A_stateTransProbs_2D, dev_B_obsEmissionProbs_2D, dev_O_obsSequence_1D, N_noOfStates, V_noOfObsSymbols, T_noOfObservations, dev_Alpha_trelis_2D, dev_probs_3D, likelyhood);
		break;
	case ComputationEnvironment::CPU:
		cudaStatus = ForwardAlgorithmCPU(dev_Pi_startProbs_1D, dev_A_stateTransProbs_2D, dev_B_obsEmissionProbs_2D, dev_O_obsSequence_1D, N_noOfStates, V_noOfObsSymbols, T_noOfObservations, dev_Alpha_trelis_2D, dev_probs_3D, likelyhood);
		break;
	}

	return cudaStatus;
}

