#include "hip/hip_runtime.h"
#include "Trellis.cuh"

#include "Utilities.h"

__host__ __device__ void TrellisScaling2D(double *host_Alpha_trelis_2D, unsigned int T_noOfObservations, unsigned int N_noOfStates, unsigned int idx_t)
{
#ifdef ROW_MAJ_ORD_MAT_ROW_FIRST_INDEX

	// determine matrix dimensions
	unsigned int dim1_Alpha = N_noOfStates;
	unsigned int dim2_Alpha = T_noOfObservations;

	// determine sum
	double sum_Alpha_t = 0;
	for (unsigned int idx_j = 0; idx_j < dim1_Alpha; idx_j++)
	{
		// determine matrix indices
		unsigned int idx_alpha_tj = idx_t*dim1_Alpha + idx_j;
		sum_Alpha_t += host_Alpha_trelis_2D[idx_alpha_tj];
	}

	if (sum_Alpha_t == 0)
		return;

	double c_t = 1.0 / sum_Alpha_t;

	// rescale alphas
	for (unsigned int idx_j = 0; idx_j < dim1_Alpha; idx_j++)
	{
		// determine matrix indices
		unsigned int idx_alpha_tj = idx_t*dim1_Alpha + idx_j;
		host_Alpha_trelis_2D[idx_alpha_tj] *= c_t;
	}

#endif
}

__host__ __device__ void TrellisInitialization2D(double *host_Alpha_trelis_2D, const double *host_Pi_startProbs_1D, const double *host_B_obsEmissionProbs_2D, const unsigned int *host_O_obsSequence_1D, int T_noOfObservations, int N_noOfStates, int V_noOfObsSymbols)
{

	// ------------------------------------------------------------------------------------------------------
	// Initialization of the Alpha_trelis
	// in the paper the initialization of the trellis is done differently, in code actually it is an initialization from the priors
	// ------------------------------------------------------------------------------------------------------

	// a_0i = pi_i --- actually data should be set up like this, but to be sure Pi is transported in an extra vector
	// alpha_0(i) = Pi_i*b_i(O_0)

	int obs_start = host_O_obsSequence_1D[0];
	// TODO: similar to the following
	//Observation observation;
	//idx_obs_T = observation.getObservationSymbolIndex(obs_start);
	// HACK: symbol id is same as index
	int idx_obs_start = obs_start;

#ifdef COL_MAJ_ORD_MAT_ROW_FIRST_INDEX

	int dim1_B = N_noOfStates;
	int dim1_Alpha = T_noOfObservations;
	// init first row of trellis
	for (unsigned int i = 0; i < N_noOfStates; i++)
	{
		int idx_b_i_idxOs = i + idx_obs_start * dim1_B;
		int idx_alpha_0i = i*dim1_Alpha;
		int idx_pi_i = i;

		double alpha_0_i = host_Pi_startProbs_1D[idx_pi_i] * host_B_obsEmissionProbs_2D[idx_b_i_idxOs];
		host_Alpha_trelis_2D[idx_alpha_0i] = alpha_0_i;
	}

#endif

#ifdef ROW_MAJ_ORD_MAT_ROW_FIRST_INDEX

	//for (i = 0; i < nstates; i++) {
	//	alpha[0][i] = prior[i] + obvs[IDX(i, data[0], nobvs)];
	//}

	// init first row of trellis
	for (unsigned int i = 0; i < N_noOfStates; i++)
	{
		int idx_b_i_idxOs = i*V_noOfObsSymbols + idx_obs_start;
		int idx_alpha_0i = i;
		int idx_pi_i = i;

		double alpha_0_i = host_Pi_startProbs_1D[idx_pi_i] * host_B_obsEmissionProbs_2D[idx_b_i_idxOs];
		host_Alpha_trelis_2D[idx_alpha_0i] = alpha_0_i;
	}

#endif
}

//---------------------------------------------------------------------------------------------------------

__host__ void AlphaTrellisSliceInitialization2D(double *host_Alpha_trelis_2D, const double *host_Pi_startProbs_1D, const double *host_B_obsEmissionProbs_2D, const unsigned int *host_O_obsSequence_1D, int N_noOfStates, int V_noOfObsSymbols, int M_noOfSequences, int idx_m)
{

	// ------------------------------------------------------------------------------------------------------
	// Initialization of the Alpha_trelis
	// in the paper the initialization of the trellis is done differently, in code actually it is an initialization from the priors
	// ------------------------------------------------------------------------------------------------------

	// a_0i = pi_i --- actually data should be set up like this, but to be sure Pi is transported in an extra vector
	// alpha_0(i) = Pi_i*b_i(O_0)

	int obs_start = host_O_obsSequence_1D[0];
	// TODO: similar to the following
	//Observation observation;
	//idx_obs_T = observation.getObservationSymbolIndex(obs_start);
	// HACK: symbol id is same as index
	int idx_obs_start = obs_start;

#ifdef ROW_MAJ_ORD_MAT_ROW_FIRST_INDEX

	// init first row of trellis
	for (unsigned int i = 0; i < N_noOfStates; i++)
	{
		int idx_b_i_idxOs = i*V_noOfObsSymbols + idx_obs_start;
		int idx_alpha_0i = idx_m*N_noOfStates + i;
		int idx_pi_i = i;

		double alpha_0_i = host_Pi_startProbs_1D[idx_pi_i] * host_B_obsEmissionProbs_2D[idx_b_i_idxOs];
		host_Alpha_trelis_2D[idx_alpha_0i] = alpha_0_i;
	}

#endif
}

__global__ void AlphaTrellisInitializationGPU(double *dev_3D_Trellis, const double *dev_Pi_startProbs_1D, const double *dev_B_obsEmissionProbs_2D, const unsigned int *dev_O_obsSequences_2D, int T_noOfObservations, int N_noOfStates, int V_noOfObsSymbols){

	int obs_index = blockIdx.x * T_noOfObservations;
	int obs_start = dev_O_obsSequences_2D[obs_index];
	int idx_b_i_idxOs = threadIdx.x*V_noOfObsSymbols + obs_start;
	int idx_alpha_0i = blockIdx.x * N_noOfStates + threadIdx.x;
	int idx_pi_i = threadIdx.x;

	double alpha_0_i = dev_Pi_startProbs_1D[idx_pi_i] * dev_B_obsEmissionProbs_2D[idx_b_i_idxOs];
	dev_3D_Trellis[idx_alpha_0i] = alpha_0_i;


}

