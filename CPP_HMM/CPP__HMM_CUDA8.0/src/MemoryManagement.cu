#include "MemoryManagement.cuh"

#include <stdio.h>
#include <stdlib.h> 
#include <string.h>

ComputationEnvironment glob_Env = ComputationEnvironment::GPU;

MemoryMovementDuplication glob_Dup = MemoryMovementDuplication::NO;

__host__ hipError_t allocateDeviceVector(IntHdl pVector, int numberOfElements, bool cleanAlloc)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		if (cleanAlloc)
		{
			cudaStatus = hipMemset(*pVector, 0, numberOfElements);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemset failed!");
				hipFree(*pVector);
				*pVector = NULL;
			}
		}
		break;
	case ComputationEnvironment::CPU:
		if (cleanAlloc)
			*pVector = (IntPtr)calloc(numberOfElements, sizeof(int));
		else
			*pVector = (IntPtr)malloc(numberOfElements * sizeof(int));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t allocateDeviceVector(UIntHdl pVector, int numberOfElements, bool cleanAlloc)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(unsigned int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		if (cleanAlloc)
		{
			cudaStatus = hipMemset(*pVector, 0, numberOfElements);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemset failed!");
				hipFree(*pVector);
				*pVector = NULL;
			}
		}
		break;
	case ComputationEnvironment::CPU:
		if (cleanAlloc)
			*pVector = (UIntPtr)calloc(numberOfElements, sizeof(unsigned int));
		else
			*pVector = (UIntPtr)malloc(numberOfElements * sizeof(unsigned int));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t allocateDeviceVector(FloatHdl pVector, int numberOfElements, bool cleanAlloc)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		if (cleanAlloc)
		{
			int factor = sizeof(float) / sizeof(int);
			cudaStatus = hipMemset(*pVector, 0, numberOfElements * factor);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemset failed!");
				hipFree(*pVector);
				*pVector = NULL;
			}
		}
		break;
	case ComputationEnvironment::CPU:
		if (cleanAlloc)
			*pVector = (float *)calloc(numberOfElements, sizeof(float));
		else
			*pVector = (float *)malloc(numberOfElements * sizeof(float));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t allocateDeviceVector(DoubleHdl pVector, int numberOfElements, bool cleanAlloc)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMalloc((void**)pVector, numberOfElements * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			hipFree(*pVector);
			*pVector = NULL;
		}
		if (cleanAlloc)
		{
			int factor = sizeof(double) / sizeof(int);
			cudaStatus = hipMemset(*pVector, 0, numberOfElements * factor);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemset failed!");
				hipFree(*pVector);
				*pVector = NULL;
			}
		}
		break;
	case ComputationEnvironment::CPU:
		if (cleanAlloc)
			*pVector = (double *)calloc(numberOfElements, sizeof(double));
		else
			*pVector = (double *)malloc(numberOfElements * sizeof(double));
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(IntPtr dst, const IntPtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(int), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		switch (glob_Dup)
		{
		case MemoryMovementDuplication::YES:
			memccpy(dst, src, numberOfElements, sizeof(int));
			cudaStatus = hipError_t::hipSuccess;
			break;
		case MemoryMovementDuplication::NO:
			dst = src;
			cudaStatus = hipError_t::hipSuccess;
			break;
		}
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(UIntPtr dst, const UIntPtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(unsigned int), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		switch (glob_Dup)
		{
		case MemoryMovementDuplication::YES:
			memccpy(dst, src, numberOfElements, sizeof(unsigned int));
			cudaStatus = hipError_t::hipSuccess;
			break;
		case MemoryMovementDuplication::NO:
			dst = src;
			cudaStatus = hipError_t::hipSuccess;
			break;
		}
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(FloatPtr dst, const FloatPtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(float), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		switch (glob_Dup)
		{
		case MemoryMovementDuplication::YES:
			memccpy(dst, src, numberOfElements, sizeof(float));
			cudaStatus = hipError_t::hipSuccess;
			break;
		case MemoryMovementDuplication::NO:
			dst = src;
			cudaStatus = hipError_t::hipSuccess;
			break;
		}
		break;
	}

	return cudaStatus;
}

__host__ hipError_t memcpyVector(DoublePtr dst, const DoublePtr src, int numberOfElements, enum hipMemcpyKind kind)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipMemcpy(dst, src, numberOfElements * sizeof(double), kind);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		break;
	case ComputationEnvironment::CPU:
		switch (glob_Dup)
		{
		case MemoryMovementDuplication::YES:
			memccpy(dst, src, numberOfElements, sizeof(double));
			cudaStatus = hipError_t::hipSuccess;
			break;
		case MemoryMovementDuplication::NO:
			dst = src;
			cudaStatus = hipError_t::hipSuccess;
			break;
		}
		break;
	}

	return cudaStatus;
}

__host__ hipError_t deviceFree(void *devPtr)
{
	hipError_t cudaStatus = hipError_t::cudaErrorIllegalInstruction;
	switch (glob_Env)
	{
	case ComputationEnvironment::GPU:
		cudaStatus = hipFree(devPtr);
		break;
	case ComputationEnvironment::CPU:
		free(devPtr);
		cudaStatus = hipError_t::hipSuccess;
		break;
	}

	return cudaStatus;
}
